
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_fp16.h>
#include <assert.h>

typedef half2 ftype;
long long sig_size = 1<<23;


int main(){

  ftype *h_idata = (ftype *)malloc(sig_size*sizeof(ftype));
  ftype *d_idata;
  ftype *d_odata;
  hipMalloc(&d_idata, sizeof(ftype)*sig_size);
  hipMalloc(&d_odata, sizeof(ftype)*sig_size);
  hipfftHandle plan;
  hipfftResult r;
  r = hipfftCreate(&plan);
  assert(r == HIPFFT_SUCCESS);
  size_t ws = 0;
  r = hipfftXtMakePlanMany(plan, 1,  &sig_size, NULL, 1, 1, HIP_C_16F, NULL, 1, 1, HIP_C_16F, 1, &ws, HIP_C_16F);
  assert(r == HIPFFT_SUCCESS);
  r = hipfftXtExec(plan, d_idata, d_odata, HIPFFT_FORWARD); // warm-up
  assert(r == HIPFFT_SUCCESS);
  hipEvent_t start, stop;
  hipEventCreate(&start); hipEventCreate(&stop);
  hipEventRecord(start);
  r = hipfftXtExec(plan, d_idata, d_odata, HIPFFT_FORWARD);
  assert(r == HIPFFT_SUCCESS);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float et;
  hipEventElapsedTime(&et, start, stop);
  printf("forward FFT time for %lld samples: %fms\n", sig_size, et);
  return 0;
}
