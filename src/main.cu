#include <hipfft/hipfft.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_fp16.h>
#include <assert.h>

/*typedef half2 ftype;*/
/*long long sig_size = 1<<23;*/


/*int main(){*/

/*  ftype *h_idata = (ftype *)malloc(sig_size*sizeof(ftype));*/
/*  ftype *d_idata;*/
/*  ftype *d_odata;*/
/*  hipMalloc(&d_idata, sizeof(ftype)*sig_size);*/
/*  hipMalloc(&d_odata, sizeof(ftype)*sig_size);*/
/*  hipfftHandle plan;*/
/*  hipfftResult r;*/
/*  r = hipfftCreate(&plan);*/
/*  assert(r == HIPFFT_SUCCESS);*/
/*  size_t ws = 0;*/
/*  r = hipfftXtMakePlanMany(plan, 1,  &sig_size, NULL, 1, 1, HIP_C_16F, NULL, 1, 1, HIP_C_16F, 1, &ws, HIP_C_16F);*/
/*  assert(r == HIPFFT_SUCCESS);*/
/*  r = hipfftXtExec(plan, d_idata, d_odata, HIPFFT_FORWARD); // warm-up*/
/*  assert(r == HIPFFT_SUCCESS);*/
/*  hipEvent_t start, stop;*/
/*  hipEventCreate(&start); hipEventCreate(&stop);*/
/*  hipEventRecord(start);*/
/*  r = hipfftXtExec(plan, d_idata, d_odata, HIPFFT_FORWARD);*/
/*  assert(r == HIPFFT_SUCCESS);*/
/*  hipEventRecord(stop);*/
/*  hipEventSynchronize(stop);*/
/*  float et;*/
/*  hipEventElapsedTime(&et, start, stop);*/
/*  printf("forward FFT time for %lld samples: %fms\n", sig_size, et);*/
/*  return 0;*/
/*}*/


#include "pinned_mapped_vector_utils.hpp"
#include "pinned_mapped_allocator.hpp"

#include "my_cufft_utils.hpp"
#include "my_cuda_utils.hpp"

#include "my_generators.hpp"
#include "my_printers.hpp"

#include "my_utils.hpp"

#include "my_cuHalfComplex.hpp"

int main(int argc, char **argv) {
   try {
      hipError_t cerror = hipSuccess;
      hipfftResult cufft_status = HIPFFT_SUCCESS;
      bool debug = false;
      
      // Empirically-determined maximum number
      long long num_vals = 1<<21;

      ////////////////////////////////////////////////////////////////////
      // ALLOCATE KERNEL DATA
      ////////////////////////////////////////////////////////////////////
      dout << "Initializing memory for input and output data...\n";
      // Allocate pinned host memory that is also accessible by the device.
      pinned_mapped_vector<cuHalfComplex> samples;
      pinned_mapped_vector<cuHalfComplex> frequencies;
      samples.reserve( num_vals );
      frequencies.reserve( num_vals );
      frequencies.resize( num_vals );

      gen_cuHalfComplexes( samples.data(), num_vals, 0.0, 1.0 );
      print_cuHalfComplexes( samples.data(), 10, "Samples", "\n", "\n" );

      hipfftHandle plan;
      size_t work_size = 0;

      try_cufft_func_throw(cufft_status, hipfftCreate(&plan) );

      try_cufft_func_throw(cufft_status, 
         hipfftXtMakePlanMany(plan, 1,  &num_vals, NULL, 1, 1, HIP_C_16F, NULL, 1, 1, HIP_C_16F, 1, &work_size, HIP_C_16F) );

      dout << "Work Size after hipfftXtMakePlanMany() is " << work_size << "\n";

      try_cufft_func_throw(cufft_status,
         hipfftXtExec(plan, samples.data(), frequencies.data(), HIPFFT_FORWARD) );

      print_cuHalfComplexes( frequencies.data(), 10, "Frequencies", "\n", "\n" );

      samples.clear();
      return SUCCESS;

   } catch( std::exception& ex ) {
      std::cout << "ERROR: " << ex.what() << "\n"; 
      return FAILURE;

   }
}
